/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Template project which demonstrates the basics on how to setup a project 
 * example application.
 * Host code.
 */

// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil_inline.h>

// includes, kernels
//#include <alignment_kernel.cu>

//********************** global variables ***************************
// BLOSUM62 matrix and its size
// these should be copied to constant memory in the device
/* const int size_blosum  = 24; */
/* int BLOSUM62[size_blosum*size_blosum] = { */
/*   4,  -1,  -2,  -2,   0,  -1,  -1,   0,  -2,  -1,  -1,  -1,  -1,  -2,  -1,   1,   0,  -3,  -2,   0,  -2,  -1,   0,  -4,    */
/*   -1,   5,   0,  -2,  -3,   1,   0,  -2,   0,  -3,  -2,   2,  -1,  -3,  -2,  -1,  -1,  -3,  -2,  -3,  -1,   0,  -1,  -4,    */
/*   -2,   0,   6,   1,  -3,   0,   0,   0,   1,  -3,  -3,   0,  -2,  -3,  -2,   1,   0,  -4,  -2,  -3,   3,   0,  -1,  -4,    */
/*   -2,  -2,   1,   6,  -3,   0,   2,  -1,  -1,  -3,  -4,  -1,  -3,  -3,  -1,   0,  -1,  -4,  -3,  -3,   4,   1,  -1,  -4,    */
/*   0,  -3,  -3,  -3,   9,  -3,  -4,  -3,  -3,  -1,  -1,  -3,  -1,  -2,  -3,  -1,  -1,  -2,  -2,  -1,  -3,  -3,  -2,  -4,    */
/*   -1,   1,   0,   0,  -3,   5,   2,  -2,   0,  -3,  -2,   1,   0,  -3,  -1,   0,  -1,  -2,  -1,  -2,   0,   3,  -1,  -4,    */
/*   -1,   0,   0,   2,  -4,   2,   5,  -2,   0,  -3,  -3,   1,  -2,  -3,  -1,   0,  -1,  -3,  -2,  -2,   1,   4,  -1,  -4,    */
/*   0,  -2,   0,  -1,  -3,  -2,  -2,   6,  -2,  -4,  -4,  -2,  -3,  -3,  -2,   0,  -2,  -2,  -3,  -3,  -1,  -2,  -1,  -4,    */
/*   -2,   0,   1,  -1,  -3,   0,   0,  -2,   8,  -3,  -3,  -1,  -2,  -1,  -2,  -1,  -2,  -2,   2,  -3,   0,   0,  -1,  -4,    */
/*   -1,  -3,  -3,  -3,  -1,  -3,  -3,  -4,  -3,   4,   2,  -3,   1,   0,  -3,  -2,  -1,  -3,  -1,   3,  -3,  -3,  -1,  -4,    */
/*   -1,  -2,  -3,  -4,  -1,  -2,  -3,  -4,  -3,   2,   4,  -2,   2,   0,  -3,  -2,  -1,  -2,  -1,   1,  -4,  -3,  -1,  -4,    */
/*   -1,   2,   0,  -1,  -3,   1,   1,  -2,  -1,  -3,  -2,   5,  -1,  -3,  -1,   0,  -1,  -3,  -2,  -2,   0,   1,  -1,  -4,    */
/*   -1,  -1,  -2,  -3,  -1,   0,  -2,  -3,  -2,   1,   2,  -1,   5,   0,  -2,  -1,  -1,  -1,  -1,   1,  -3,  -1,  -1,  -4,    */
/*   -2,  -3,  -3,  -3,  -2,  -3,  -3,  -3,  -1,   0,   0,  -3,   0,   6,  -4,  -2,  -2,   1,   3,  -1,  -3,  -3,  -1,  -4,    */
/*   -1,  -2,  -2,  -1,  -3,  -1,  -1,  -2,  -2,  -3,  -3,  -1,  -2,  -4,   7,  -1,  -1,  -4,  -3,  -2,  -2,  -1,  -2,  -4,    */
/*   1,  -1,   1,   0,  -1,   0,   0,   0,  -1,  -2,  -2,   0,  -1,  -2,  -1,   4,   1,  -3,  -2,  -2,   0,   0,   0,  -4,    */
/*   0,  -1,   0,  -1,  -1,  -1,  -1,  -2,  -2,  -1,  -1,  -1,  -1,  -2,  -1,   1,   5,  -2,  -2,   0,  -1,  -1,   0,  -4,    */
/*   -3,  -3,  -4,  -4,  -2,  -2,  -3,  -2,  -2,  -3,  -2,  -3,  -1,   1,  -4,  -3,  -2, 11,   2,  -3,  -4,  -3,  -2,  -4,    */
/*   -2,  -2,  -2,  -3,  -2,  -1,  -2,  -3,   2,  -1,  -1,  -2,  -1,   3,  -3,  -2,  -2,   2,   7,  -1,  -3,  -2,  -1,  -4,    */
/*   0,  -3,  -3,  -3,  -1,  -2,  -2,  -3,  -3,   3,   1,  -2,   1,  -1,  -2,  -2,   0,  -3,  -1,   4,  -3,  -2,  -1,  -4,    */
/*   -2,  -1,   3,   4,  -3,   0,   1,  -1,   0,  -3,  -4,   0,  -3,  -3,  -2,   0,  -1,  -4,  -3,  -3,   4,   1,  -1,  -4,    */
/*   -1,   0,   0,   1,  -3,   3,   4,  -2,   0,  -3,  -3,   1,  -1,  -3,  -1,   0,  -1,  -3,  -2,  -2,   1,   4,  -1,  -4,    */
/*   0,  -1,  -1,  -1,  -2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -2,   0,   0,  -2,  -1,  -1,  -1,  -1,  -1,  -4,    */
/*   -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,  -4,   1 */
/* }; */

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void alignment();

// external definition of the alignment kernel
extern "C"
void alignmentKernel( int* g_idata, int* g_index, int* g_odata );



////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
//void runTest( int argc, char** argv) 
void alignment()
{
  /* // use command-line specified CUDA device, otherwise use device with highest Gflops/s */
  /* if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) */
  /*   cutilDeviceInit(argc, argv); */
  /* else */
  /*   cudaSetDevice( cutGetMaxGflopsDeviceId() ); */

  /* // setup the timer */
  /* unsigned int timer = 0;  */
  /* cutilCheckError( cutCreateTimer( &timer)); */
  /* cutilCheckError( cutStartTimer( timer)); */

  /* // input */
  /* char *a = "atcggtcaattggcc"; */
  /* char *b = "atgccctgatgggccga"; */

  /* // dimension of multi-threading */
  /* unsigned int num_threads = 1; */
  /* unsigned int mem_size = sizeof( char ) * (num_threads + 1) * 1000; */
  /* unsigned int index_size = (num_threads + 2) * sizeof(int); */
  /* unsigned int out_size = num_threads * sizeof(int); */
  /* unsigned int shared_mem_size =  num_threads * strlen(a) * sizeof(char) * 2 ; */

  /* // allocate host memory */
  /* char* h_idata = (char*) malloc( mem_size ); */
  /* int* h_index = (int*) malloc( index_size ); */
  /* // initalize the memory */
  /* /\* for( unsigned int i = 0; i < num_threads; ++i)  */
  /*    { */
  /*    h_idata[i] = (float) i; */
  /*    }*\/ */
	
  /* *(h_index) = strlen(a); */
  /* *(h_index+1) = strlen(a) + strlen(b); */
  /* memcpy( h_idata, a, strlen(a) * sizeof(char) ); */
  /* memcpy( h_idata + strlen(a) * sizeof(char), b, strlen(b) * sizeof(char) ); */

  /* // allocate device memory */
  /* int* d_idata; */
  /* int* d_index; */
  /* cutilSafeCall( cudaMalloc( (void**) &d_idata, mem_size)); */
  /* cutilSafeCall( cudaMalloc( (void**) &d_index, index_size ) ); */
  /* // copy host memory to device */
  /* cutilSafeCall( cudaMemcpy( d_idata, h_idata, mem_size, cudaMemcpyHostToDevice) ); */
  /* cutilSafeCall( cudaMemcpy( d_index, h_index, index_size, cudaMemcpyHostToDevice ) ); */

  /* // allocate device memory for result */
  /* int* d_odata; */
  /* cutilSafeCall( cudaMalloc( (void**) &d_odata, out_size ) ); */

  /* // setup execution parameters */
  /* dim3  grid( 1, 1, 1); */
  /* dim3  threads( num_threads, 1, 1); */

  /* // execute the kernel */
  /* alignmentKernel<<< grid, threads, shared_mem_size >>>( d_idata, d_index,d_odata); */

  /* // check if kernel execution generated and error */
  /* cutilCheckMsg("Kernel execution failed"); */

  /* // allocate mem for the result on host side */
  /* int* h_odata = (int*) malloc( out_size ); */
  /* // copy result from device to host */
  /* cutilSafeCall( cudaMemcpy( h_odata, d_odata, sizeof( float) * num_threads, */
  /* 			     cudaMemcpyDeviceToHost) ); */

  /* // compute the time spent for device computation */
  /* cutilCheckError( cutStopTimer( timer)); */
  /* printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer)); */
  /* cutilCheckError( cutDeleteTimer( timer)); */

  /* // calculate the final result */
  /* int result = 0; */
  /* for ( int i =0; i< num_threads; ++i ) */
  /*   { */
  /*     result += *( h_odata + i ); */
  /*   } */

  /* printf("result is %d\n", result); */

  /* // cleanup memory */
  /* free( h_idata); */
  /* free( h_index ); */
  /* free( h_odata); */
  /* cutilSafeCall(cudaFree(d_idata)); */
  /* cutilSafeCall(cudaFree(d_odata)); */
  /* cutilSafeCall(cudaFree(d_index)); */

  /* cudaThreadExit(); */
}
