#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* alignment project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code.
 */

#ifndef _ALIGNMENT_KERNEL_H_
#define _ALIGNMENT_KERNEL_H_

#include <stdio.h>

#define MAX_SHARED_MEM 1000;
//#define SDATA( index)      cutilBankChecker(sdata, index)

/**
 *\brief init_sequence
 *
 * Initialize the sequences in device space
 */
__global__ void init_sequence()
{

}


__device__ inline int score(char ch1, char ch2)
{
  return (ch1 == '-' || ch2 == '-')? 0 : 2;
}



__global__ void alignmentKernel( char* query, int query_size, char* seqbase, int* seqbase_idx, int* score_list )
{
  // each thread compute a single alignment from the database
  int idx = threadIdx.x + blockIdx.x * blockDim.x ;

  // the host will determine the number of threads in each block
  // by the buffer size and query size
  char* sequence = seqbase + seqbase_idx[idx];
  int seq_size = seqbase_idx[idx+1] - seqbase[idx];
  
  // store the alignment buffer in the shared memory
  // each alignment will use a single buffer
  // whose size is the same as the length of the query
  // TODO: resolve the bank conflict issue
  __shared__ int g_buffer[MAX_SHARED_MEM];
  for (int i=0; i<seq_size; g_buffer[threadIdx.x * seq_size + i] = 0, ++i);

  int* buffer = g_buffer[threadIdx.x * seq_size];
  int max_val = -1;
  for ( int i = 0; i < seq_size; ++i)
    {
      int prev = 0;
      for ( int j = 1; j < query_size; ++j )
  	{
  	  char ch1 = sequence[i];
  	  char ch2 = query[j];
  	  int vl = prev + score(ch1, '-');
  	  int vu = buffer[j] + score(ch1, '-');
  	  int vd = buffer[j-1] + score(ch1, ch2);
  	  buffer[j-1] = prev;
  	  prev = max(vl, max(vd, vu));
  	  max_val = max(max_val, prev);
  	}
    }
  score_list[idx] = max_val;
}



#endif // #ifndef _alignment_KERNEL_H_
